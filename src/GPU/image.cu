#include "hip/hip_runtime.h"
#include "image.hh"
#include <cstdlib>
#include <iostream>
#include <math.h>
#include <spdlog/spdlog.h>
#include <stdio.h>

#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0); \
 }                                                                 \
}

[[gnu::noinline]]
void _abortError(const char* msg, const char* fname, int line)
{
  hipError_t err = hipGetLastError();
  std::cout << "error : " << line << msg << std::endl;
  spdlog::error("{} ({}, line: {})", msg, fname, line);
  spdlog::error("Error {}: {}", hipGetErrorName(err), hipGetErrorString(err));
  std::exit(1);
}

#define abortError(msg) _abortError(msg, __FUNCTION__, __LINE__)
/*__global__ void kvecPrint(double *img, int lgt) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= lgt)
	return;
    res_img[i] = img1[i] + img2[i];
}*/

__host__ gray8_image::gray8_image(int height, int width, png_bytep *row_pointers) {
    sx = height;
    sy = width;
    length = sx * sy;

    auto rc = hipMallocManaged(&pixels, sizeof(double) * length);//new double[length];

    if (rc)
        abortError("Fail buffer allocation in gray8_image");

    double *buffer = new double[length];
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            png_bytep pixel = &(row_pointers[i][j * 4]);
            auto r = pixel[0];
            auto g = pixel[1];
            auto b = pixel[2];
            auto transp = pixel[3];
            buffer[i * width + j] = 0.299 * r + 0.587 * g + 0.114 * b;
        }
    }
    get_data_from(buffer);
    /*for (int i = 0; i < length; i++)
	    std::cout << pixels[i] << " | ";
    std::cout << std::endl;*/
}

__host__ gray8_image::gray8_image(int _sx, int _sy) {
    this->sx = _sx;
    this->sy = _sy;

    this->length = sx * sy;
    //this->pixels = new double[length];
    auto rc = hipMallocManaged(&pixels, sizeof(double) * length);//new double[length];
    if (rc)
        abortError("Fail buffer allocation in gray8_image");
}

__host__ gray8_image::~gray8_image() {
    auto rc = hipFree(pixels);
    if (rc)
        abortError("Fail buffer free in gray8_image");
    //delete [] pixels;
}

__host__ void gray8_image::get_data_from(double *input) {
    std::cout << "Starting hipMemcpy..." << std::endl;
    hipMemcpy(pixels, input, length * sizeof(double), hipMemcpyHostToDevice);
    //std::cout << "Starting synchronize..." << std::endl;
    //hipDeviceSynchronize();
}

__device__ double *&gray8_image::get_buffer() {
    return pixels;
}

__global__ void kvecAdd(double *img1, double *img2, double *res_img, int lgt) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= lgt)
	return;
    res_img[i] = img1[i] + img2[i];
}

/*__device__ gray8_image *img_add(gray8_image *img, gray8_image *img2, gray8_image *res_img) {
    dim3 dimBlock(32, 32);
    dim3 dimGrid(img->sx/dimBlock.x, img->sy/dimBlock.y);
    kvecAdd<<<dimBlock,dimGrid>>>(img->pixels, img2->pixels, res_img->pixels, img->length);
    return res_img;
}*/

__global__ void kvecMultScalar(double *img1, int val, double *res_img, int lgt) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= lgt)
	return;
    res_img[i] = img1[i] * val;
}


__global__ void kvecMult(double *img1, double *img2, double *res_img, int lgt) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= lgt)
	return;
    res_img[i] = img1[i] * img2[i];
    /*if (i > 500) {
    printf("i : %d, img1[i]:%lf\n", i, img1[i]);
    printf("i : %d, img2[i]:%lf\n", i, img2[i]);
    printf("res_img[i]:%lf\n",res_img[i]);
    }*/
}



__global__ void kvecDiv(double *img1, double *img2, double *res_img, int lgt) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= lgt)
	return;
    res_img[i] = img1[i] / img2[i];
}


__global__ void kvecSous(double *img1, double *img2, double *res_img, int lgt) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= lgt)
	return;
    res_img[i] = img1[i] - img2[i];
}


__global__ void kvecAddScalar(double *img1, int value, double *res_img, int lgt) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= lgt)
	return;
    res_img[i] = img1[i] + value;
}


__global__ void kvecConvol(double *img, int img_x, int img_y, double *mask, int msk_size, double *res_img) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    if (x >= img_x)
	return;
    if (y >= img_y)
	return;
    //printf("x: %d, y: %d | index = %d\n", x, y, x * img_y + y);

    int index = (msk_size - 1) / 2;
    double res = 0;
    for (int i = -index; i <= index; i++) {
        if (i + x < 0 || i + x >= img_x) {
            continue;
        }
        for (int j = -index; j <= index; j++) {
            if (j + y < 0 || j + y >= img_y) {
                continue;
            }

            double m = mask[(i + index) * msk_size + (j + index)];
            double n = img[(x + i) * img_y + (y + j)];
            res += m * n;
        }
    }
    res_img[x * img_y + y] = res;
}

/*
void gray8_image::gray_convolution(gray8_image* masque, gray8_image* res_img) {
    int index = (masque->sx - 1) / 2;
    for (int x = 0; x < this->sx; x++) {
        for (int y = 0; y < this->sy; y++) {
            double res = 0;
            for (int i = -index; i <= index; i++) {
                if (i + x < 0 || i + x >= this->sx) {
                    continue;
                }
                for (int j = -index; j <= index; j++) {
                    if (j + y < 0 || j + y >= this->sy) {
                        continue;
                    }

                    double m = masque->pixels[(i + index) * masque->sy + (j + index)];
                    double n = this->pixels[(x + i) * this->sy + (y + j)];
                    res += m * n;
                }
            }
            res_img->pixels[x * this->sy + y] = res;
        }
    }
}
*/
__global__ void kvecDilate(double *img, int img_x, int img_y, double *mask, int msk_size, double *res_img) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    if (x >= img_x)
	return;
    if (y >= img_y)
	return;
    //printf("x: %d, y: %d | index = %d\n", x, y, x * img_y + y);

    int index = (msk_size - 1) / 2;
    double max = img[x * img_y + y];
    for (int i = -index; i <= index; i++) {
        if (i + x < 0 || i + x >= img_x) {
            continue;
        }
        for (int j = -index; j <= index; j++) {
            if (j + y < 0 || j + y >= img_y) {
                continue;
            }

            double m = mask[(i + index) * msk_size + (j + index)];
	    if (m == 0)
		continue;
            double n = img[(x + i) * img_y + (y + j)];
	    if (n > max)
		max = n;
        }
    }
    res_img[x * img_y + y] = max;
}

gray8_image *gray8_image::dilate(gray8_image* masque) {
    int index = (masque->sx - 1) / 2;
    gray8_image *res_img = new gray8_image(this->sx, this->sy);
    for (int x = 0; x < this->sx; x++) {
        for (int y = 0; y < this->sy; y++) {
            double max = this->pixels[x * this->sy + y];
            for (int i = -index; i <= index; i++) {
                if (i + x < 0 || i + x >= this->sx) {
                    continue;
                }
                for (int j = -index; j <= index; j++) {
                    if (j + y < 0 || j + y >= this->sy) {
                        continue;
                    }

                    double m = masque->pixels[(i + index) * masque->sy + (j + index)];
                    if (m == 0) {
                        continue;
                    }
                    double n = this->pixels[(x + i) * this->sy + (y + j)];
                    if (n > max) {
                        max = n;
                    }
                }
            }
            res_img->pixels[x * this->sy + y] = max;
        }
    }
    return res_img;
}

float gray8_image::max() {
    float res = this->pixels[0];
    for (int i = 0; i < this->sx; i++) {
        for (int j = 0; j < this->sy; j++) {
            if (this->pixels[i * this->sy + j] > res) {
                res = this->pixels[i * this->sy + j];
            }
        }
    }
    return res;
}

float gray8_image::min() {
    float res = this->pixels[0];
    for (int i = 0; i < this->sx; i++) {
        for (int j = 0; j < this->sy; j++) {
            if (this->pixels[i * this->sy + j] < res) {
                res = this->pixels[i * this->sy + j];
            }
        }
    }
    return res;
}
